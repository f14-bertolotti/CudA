#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <cstdint>
#include <cstdlib>
#include <hipfft/hipfft.h>
#include <math.h>

#include "../utils/utils.cu"

#define GSIZE 1024
#define KSIZE 11
#define STATES 13
#define TIME 100
#define SEED 1

hipfftReal* get_random_grid(int states, int size, int seed) {
    srand(seed);
    hipfftReal* grid = (hipfftReal*) malloc(sizeof(hipfftReal)*size*size);
    for (int i = 0; i < size * size; ++i) grid[i] = (float) (rand() % states); 

    return grid;
}

hipfftReal* get_circular_kernel(int states, int ksize, int gsize) {
    hipfftReal* kernel = (hipfftReal*) calloc(gsize*gsize, sizeof(hipfftReal));
    int total = 0;
    for(int i = -ksize/2; i < (ksize%2 ? ksize/2+1 : ksize/2); ++i) {
        for(int j = -ksize/2; j < (ksize%2 ? ksize/2+1 : ksize/2); ++j) {
             int l = pow(i,2) + pow(j,2);
             if (6 < l && l < 30) {
                 kernel[(i>=0?i:gsize+i)*gsize+(j>=0?j:gsize+j)] = 1.0f;
                 ++total;
             }
        }
    }
    for (int i = 0; i < gsize * gsize; ++i) 
        kernel[i] = kernel[i] / (total * (states-1));
 
    return kernel;
}

__global__ void multistate_gol_growth(hipfftReal* grid, hipfftReal* neigh, int states, int time, int size) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    hipfftReal u = neigh[id];
    hipfftReal a = grid[id];
    grid[id] = min((hipfftReal) states-1,max(0.0f,a + (1.0f/time)*(((u>=0.12f)&(u<=0.15)) - ((u<0.12)|(u>=0.15)))));
}

__global__ void colorize(uint8_t* color_field, hipfftReal* grid, int states, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    uint8_t v = (uint8_t) round(grid[i]);
    color_field[4 * i + 0] = v * 255.0f/states;
	color_field[4 * i + 1] = v * 255.0f/states;
	color_field[4 * i + 2] = v * 255.0f/states;
	color_field[4 * i + 3] = 255;
}

int main (int argc, char *argv[]) { 

    sf::RenderWindow window(sf::VideoMode(GSIZE, GSIZE), "larger than life fft");
    sf::Texture texture;
	sf::Sprite sprite;
	std::vector<sf::Uint8> pixelBuffer(GSIZE * GSIZE * 4);
	texture.create(GSIZE, GSIZE);

    hipfftHandle planR2C;
    hipfftHandle planC2R;

    hipfftPlan2d(&planR2C, GSIZE, GSIZE, HIPFFT_R2C);
    hipfftPlan2d(&planC2R, GSIZE, GSIZE, HIPFFT_C2R);
 
    hipfftReal* kernel = get_circular_kernel(STATES, KSIZE, GSIZE);
    hipfftReal* grid   = get_random_grid(STATES, GSIZE, SEED);

    uint8_t*      device_color_field    = NULL;
    hipfftReal*    device_real_grid      = NULL;
    hipfftReal*    device_real_neigh     = NULL;
    hipfftReal*    device_real_kernel    = NULL;
    hipfftComplex* device_complex_grid   = NULL;
    hipfftComplex* device_complex_kernel = NULL;
    
    hipMalloc(&device_real_grid     , sizeof(hipfftReal)    * GSIZE * GSIZE);
    hipMalloc(&device_real_neigh    , sizeof(hipfftReal)    * GSIZE * GSIZE);
    hipMalloc(&device_real_kernel   , sizeof(hipfftReal)    * GSIZE * GSIZE);
    hipMalloc(&device_complex_grid  , sizeof(hipfftComplex) * GSIZE * GSIZE);
    hipMalloc(&device_complex_kernel, sizeof(hipfftComplex) * GSIZE * GSIZE);
    hipMalloc(&device_color_field   , sizeof(uint8_t) * 4  * GSIZE * GSIZE);

    hipMemcpy(device_real_grid  , grid  , sizeof(hipfftReal) * GSIZE * GSIZE, hipMemcpyHostToDevice);
    hipMemcpy(device_real_kernel, kernel, sizeof(hipfftReal) * GSIZE * GSIZE, hipMemcpyHostToDevice);
    
    hipfftExecR2C(planR2C, device_real_kernel, device_complex_kernel);

    float avg_clock = 0.0f;
    for (int n = 0;; ++n) {
        clock_t start = clock();

        hipfftExecR2C(planR2C, device_real_grid, device_complex_grid);
        scaled_hadamart_product<<<dim3(GSIZE*GSIZE/256),dim3(256)>>>(device_complex_grid, device_complex_kernel, GSIZE);
        hipfftExecC2R(planC2R, device_complex_grid, device_real_neigh);
        multistate_gol_growth<<<dim3(GSIZE*GSIZE/256), dim3(256)>>>(device_real_grid, device_real_neigh, STATES, TIME, GSIZE);
        colorize<<<dim3(GSIZE*GSIZE/256), dim3(256)>>>(device_color_field, device_real_grid, STATES, GSIZE);

        hipMemcpy(pixelBuffer.data(), device_color_field, sizeof(uint8_t)*GSIZE*GSIZE*4, hipMemcpyDeviceToHost);
        
        texture.update(pixelBuffer.data());
		sprite.setTexture(texture);
		sprite.setScale({2, 2});
		window.draw(sprite);
		window.display();

        int msec = ((clock() - start) * 1000 / CLOCKS_PER_SEC)%1000;
        avg_clock = (msec + (n) * avg_clock) / (n+1);
        printf("\rmsec: %03d, avg:%3.5f.", msec, avg_clock);
        fflush(stdout);
    }

    hipfftDestroy(planR2C);
    hipfftDestroy(planC2R);

    hipFree(device_real_grid);
    hipFree(device_real_neigh);
    hipFree(device_real_kernel);
    hipFree(device_complex_grid);
    hipFree(device_complex_kernel);
    hipFree(device_color_field);



    free(grid);
    free(kernel);
}
