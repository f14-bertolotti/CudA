#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <cstdint>
#include <cstdlib>
#include <hipfft/hipfft.h>

#include "../utils/utils.cu"

#define GSIZE 2048
#define STATES 13
#define TIME 10
#define SEED 1


hipfftReal* get_random_grid(int states, int size, int seed) {
    srand(seed);
    hipfftReal* grid = (hipfftReal*) malloc(sizeof(hipfftReal)*size*size);
    for (int i = 0; i < size * size; ++i) grid[i] = rand() % states; 
    return grid;
}

hipfftReal* get_kernel(int states, int size) {
    hipfftReal* kernel = (hipfftReal*) calloc(size*size, sizeof(hipfftReal));
    kernel[1]               = 1.0f/(8*(states-1));
    kernel[size-1]          = 1.0f/(8*(states-1));
    kernel[size]            = 1.0f/(8*(states-1));
    kernel[size+1]          = 1.0f/(8*(states-1));
    kernel[2*size-1]        = 1.0f/(8*(states-1));
    kernel[size*(size-1)]   = 1.0f/(8*(states-1));
    kernel[size*(size-1)+1] = 1.0f/(8*(states-1));
    kernel[size*size-1]     = 1.0f/(8*(states-1));
    return kernel;
}

__global__ void multistate_gol_growth(hipfftReal* grid, hipfftReal* neigh, int states, int time, int size) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    hipfftReal u = neigh[id];
    hipfftReal a = grid[id];
    grid[id] = min((hipfftReal) states-1,max(0.0f,a + (1.0f/time)*(((u>=0.20f)&(u<=0.25)) - ((u<=0.18)|(u>=0.33)))));
}

__global__ void colorize(uint8_t* color_field, hipfftReal* grid, int states, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    uint8_t v = (uint8_t) round(grid[i]);
    color_field[4 * i + 0] = v * 255.0f/states;
	color_field[4 * i + 1] = v * 255.0f/states;
	color_field[4 * i + 2] = v * 255.0f/states;
	color_field[4 * i + 3] = 255;
}



int main (int argc, char *argv[]) { 
    sf::RenderWindow window(sf::VideoMode(GSIZE, GSIZE), "larger than life fft");
    sf::Texture texture;
	sf::Sprite sprite;
	std::vector<sf::Uint8> pixelBuffer(GSIZE * GSIZE * 4);
	texture.create(GSIZE, GSIZE);

    hipfftHandle planR2C;
    hipfftHandle planC2R;

    hipfftPlan2d(&planR2C, GSIZE, GSIZE, HIPFFT_R2C);
    hipfftPlan2d(&planC2R, GSIZE, GSIZE, HIPFFT_C2R);
    
    hipfftReal*   grid    = get_random_grid(STATES, GSIZE, SEED);
    hipfftReal* kernel    = get_kernel(STATES, GSIZE);

    uint8_t*      device_color_field    = NULL;
    hipfftReal*    device_real_grid      = NULL;
    hipfftReal*    device_real_neigh     = NULL;
    hipfftReal*    device_real_kernel    = NULL;
    hipfftComplex* device_complex_grid   = NULL;
    hipfftComplex* device_complex_kernel = NULL;
    
    hipMalloc(&device_real_grid     , sizeof(hipfftReal)    * GSIZE * GSIZE);
    hipMalloc(&device_real_neigh    , sizeof(hipfftReal)    * GSIZE * GSIZE);
    hipMalloc(&device_real_kernel   , sizeof(hipfftReal)    * GSIZE * GSIZE);
    hipMalloc(&device_complex_grid  , sizeof(hipfftComplex) * GSIZE * GSIZE);
    hipMalloc(&device_complex_kernel, sizeof(hipfftComplex) * GSIZE * GSIZE);
    hipMalloc(&device_color_field   , sizeof(uint8_t) * 4  * GSIZE * GSIZE);

    hipMemcpy(device_real_grid  , grid  , sizeof(hipfftReal) * GSIZE * GSIZE, hipMemcpyHostToDevice);
    hipMemcpy(device_real_kernel, kernel, sizeof(hipfftReal) * GSIZE * GSIZE, hipMemcpyHostToDevice);
    
    hipfftExecR2C(planR2C, device_real_kernel, device_complex_kernel);

    float avg_clock = 0.0f;
    for (int n = 0;; ++n) {
        clock_t start = clock();


        hipfftExecR2C(planR2C, device_real_grid, device_complex_grid);
        scaled_hadamart_product<<<dim3(GSIZE*GSIZE/256),dim3(256)>>>(device_complex_grid, device_complex_kernel, GSIZE);
        hipfftExecC2R(planC2R, device_complex_grid, device_real_neigh);
        multistate_gol_growth<<<dim3(GSIZE*GSIZE/256), dim3(256)>>>(device_real_grid, device_real_neigh, STATES, TIME, GSIZE);
        colorize<<<dim3(GSIZE*GSIZE/256), dim3(256)>>>(device_color_field, device_real_grid, STATES, GSIZE);

        hipMemcpy(pixelBuffer.data(), device_color_field, sizeof(uint8_t)*GSIZE*GSIZE*4, hipMemcpyDeviceToHost);
        
        texture.update(pixelBuffer.data());
		sprite.setTexture(texture);
		sprite.setScale({2, 2});
		window.draw(sprite);
		window.display();

        int msec = ((clock() - start) * 1000 / CLOCKS_PER_SEC)%1000;
        avg_clock = (msec + (n) * avg_clock) / (n+1);
        printf("\rmsec: %03d, avg:%3.5f.", msec, avg_clock);
        fflush(stdout);
    }

    hipfftDestroy(planR2C);
    hipfftDestroy(planC2R);

    hipFree(device_real_grid);
    hipFree(device_real_neigh);
    hipFree(device_real_kernel);
    hipFree(device_complex_grid);
    hipFree(device_complex_kernel);
    hipFree(device_color_field);

    free(grid);
    free(kernel);
    return 0;
}
