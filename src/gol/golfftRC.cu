#include "hip/hip_runtime.h"
#include <SFML/Graphics/PrimitiveType.h>
#include <SFML/Graphics/RenderWindow.h>
#include <SFML/Graphics/Types.h>
#include <SFML/Graphics/VertexArray.h>
#include <SFML/Window/VideoMode.h>
#include <SFML/Window/Window.h>
#include <time.h>
#include <cstdlib>
#include <hipfft/hipfft.h>
#include <stdio.h>

#define SIZE 1024
#define SEED 1
#define CBYTES SIZE*SIZE*sizeof(hipfftComplex)
#define FBYTES SIZE*SIZE*sizeof(hipfftReal)

hipfftComplex* getGOLKernelFFT() {

    // define host game of life kernel
    hipfftReal* hgrid = (hipfftReal*) calloc(SIZE*SIZE, sizeof(hipfftReal));
    hgrid[1]               = 1.0;
    hgrid[SIZE-1]          = 1.0;
    hgrid[SIZE]            = 1.0;
    hgrid[SIZE+1]          = 1.0;
    hgrid[2*SIZE-1]        = 1.0;
    hgrid[SIZE*(SIZE-1)]   = 1.0;
    hgrid[SIZE*(SIZE-1)+1] = 1.0;
    hgrid[SIZE*SIZE-1]     = 1.0;

    // copy host kernel to device
    hipfftComplex* cgrid;
    hipfftReal*    rgrid;
    hipMalloc(&rgrid, FBYTES);
    hipMemcpy(rgrid, hgrid, FBYTES, hipMemcpyHostToDevice);
    hipMalloc(&cgrid, CBYTES);
    hipMemset(cgrid, 0, CBYTES);

    // run fft on device
    hipfftHandle planR2C;
    hipfftPlan2d(&planR2C, SIZE, SIZE, HIPFFT_R2C);
    hipfftExecR2C(planR2C, rgrid, cgrid);

    // free resources
    hipfftDestroy(planR2C);
    hipFree(rgrid);
    free(hgrid);

    return cgrid;
}

hipfftReal* getGOLGrid() {
    // get random host grid
    srand(SEED);
    hipfftReal* hgrid = (hipfftReal*) calloc(SIZE*SIZE, sizeof(hipfftReal));
    for (int i = 0; i < SIZE*SIZE; ++i) hgrid[i] = rand() % 2;

    // copy host grid to device
    hipfftReal* dgrid;
    hipMalloc(&dgrid, CBYTES);
    hipMemcpy(dgrid, hgrid, CBYTES, hipMemcpyHostToDevice);

    // free host resources
    free(hgrid);

    return dgrid;
 }

__global__ void emmul(hipfftComplex* A, hipfftComplex* B) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float x = (A[i].x*B[i].x - A[i].y*B[i].y);
    float y = (A[i].x*B[i].y + A[i].y*B[i].x);
    A[i].x = x/(SIZE*SIZE);
    A[i].y = y/(SIZE*SIZE);
}

__global__ void growth(hipfftReal* neighbours, hipfftReal* grid) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int n = round(neighbours[i]);
    int c = round(grid[i]);
    grid[i] = max(0, min(1, c + 0 + (n == 3) - ((n < 2) || (n > 3))));
}


int main(int argc, char* argv[]) {

    // create window
    sfRenderWindow* window = sfRenderWindow_create((sfVideoMode){1000, 1000, 32}, "game of life", sfResize | sfClose, NULL);
    if (!window) return EXIT_FAILURE;
    
    // create vertex buffer
    sfVertexArray* vertex_array = sfVertexArray_create();
    sfVertexArray_setPrimitiveType(vertex_array, sfPoints);
    for (int i = 0; i < SIZE*SIZE; ++i) {
        sfVertex vertex;
        vertex.color = sfBlack;
        vertex.position = (sfVector2f){(float) i / SIZE, (float) (i % SIZE)};
        sfVertexArray_append(vertex_array, vertex);
    }

    // init game of life and kernel
    hipfftReal*    hgrid  = (hipfftReal*)   calloc(SIZE*SIZE, sizeof(hipfftReal));
    hipfftComplex* cgrid  = (hipfftComplex*)calloc(SIZE*SIZE, sizeof(hipfftComplex));
    hipfftComplex* kernel = getGOLKernelFFT();
    hipfftReal*    grid   = getGOLGrid();
    
    hipfftComplex* cneigh = NULL;
    hipfftReal*    rneigh = NULL;
    hipMalloc(&rneigh, sizeof(hipfftReal   )*SIZE*SIZE);
    hipMalloc(&cneigh, sizeof(hipfftComplex)*SIZE*SIZE);
    hipMemset(rneigh, 0, FBYTES);
    hipMemset(cneigh, 0, CBYTES);

    hipfftHandle planR2C;
    hipfftHandle planC2R;
    hipfftPlan2d(&planR2C, SIZE, SIZE, HIPFFT_R2C);
    hipfftPlan2d(&planC2R, SIZE, SIZE, HIPFFT_C2R);

    // main loop
    float avg_clock = 0;
    for(int n = 0;; ++n) {
        // start timer
        clock_t start = clock();

        hipfftExecR2C(planR2C, grid, cneigh);
        emmul<<<dim3(SIZE*8), dim3(128)>>> (cneigh, kernel);
        hipfftExecC2R(planC2R, cneigh, rneigh);
        growth<<<dim3(SIZE*8), dim3(128)>>> (rneigh, grid);

        for (int i = 0; i < SIZE*SIZE; ++i) {
            sfVertex* vertex = sfVertexArray_getVertex(vertex_array, i);
            vertex->color = round(hgrid[i]) ? sfWhite : sfBlack;
        }

        sfRenderWindow_drawVertexArray(window, vertex_array, NULL);
        sfRenderWindow_display(window);

        hipMemcpy(hgrid, grid, FBYTES, hipMemcpyDeviceToHost);

        // take time
        int msec = ((clock() - start) * 1000 / CLOCKS_PER_SEC)%1000;
        if(n > 100) avg_clock = (msec + (n-100) * avg_clock) / (n+1-100);
        printf("\rmsec: %d, avg:%f.", msec, avg_clock);
        fflush(stdout);
    }


    hipfftDestroy(planR2C);
    hipfftDestroy(planC2R);
    hipFree(kernel);
    hipFree(grid);
    hipFree(rneigh);
    hipFree(cneigh);
    free(hgrid);

    return 0;
}

