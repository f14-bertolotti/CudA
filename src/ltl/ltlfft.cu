#include "hip/hip_runtime.h"
#include <SFML/Graphics/PrimitiveType.h>
#include <SFML/Graphics/RenderWindow.h>
#include <SFML/Graphics/Types.h>
#include <SFML/Graphics/VertexArray.h>
#include <SFML/Window/VideoMode.h>
#include <SFML/Window/Window.h>

#include <time.h>
#include <hipfft/hipfft.h>
#include <stdio.h>

#include "../utils/utils.cu"

#define GSIZE 1024
#define KSIZE 11 
#define RBYTES GSIZE*GSIZE*sizeof(hipfftReal)

#define BOSCO_B1 34
#define BOSCO_B2 45
#define BOSCO_S1 34
#define BOSCO_S2 58


hipfftReal* get_kernel_in_grid(int ksize, int gsize) {
    hipfftReal* hgrid = (hipfftReal*) calloc(sizeof(hipfftReal), gsize*gsize);

    for(int i = -ksize/2; i < (ksize%2 ? ksize/2+1 : ksize/2); ++i)
        for(int j = -ksize/2; j < (ksize%2 ? ksize/2+1 : ksize/2); ++j)
            hgrid[(i>=0?i:gsize+i)*gsize+(j>=0?j:gsize+j)] = 1;

    return hgrid;
}

hipfftReal* get_u10_grid(int gsize, int seed) {
    srand(seed);
    hipfftReal* hgrid = (hipfftReal*) calloc(sizeof(hipfftReal), gsize*gsize);

    for (int i = 0; i < gsize*gsize; ++i)
        hgrid[i] = rand()%2;

    return hgrid;
}


void device_rfft2(hipfftReal* in, hipfftComplex* out, int size) {
    hipfftHandle plan;
    hipfftPlan2d(&plan, size, size, HIPFFT_R2C);
    hipfftExecR2C(plan, in, out);
    hipfftDestroy(plan);
}

void device_irfft2(hipfftComplex* in, hipfftReal* out, int size) {
    hipfftHandle plan;
    hipfftPlan2d(&plan, size, size, HIPFFT_C2R);
    hipfftExecC2R(plan, in, out);
    hipfftDestroy(plan);
}

__global__ void bosco_growth(hipfftReal* grid, hipfftReal* neigh, int b1, int b2, int s1, int s2, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int c = grid[i];
    int n = neigh[i];
    grid[i] = max(0,min(1, c + ((n >= b1) & (n <= b2)) - ((n < s1) | (n > s2))));
}

int main(int argc, char* argv[]) {

    int seed = time(0);
    printf("seed: %d;\n", seed);

    sfRenderWindow* window = sfRenderWindow_create((sfVideoMode){1000, 1000, 32}, "game of life", sfResize | sfClose, NULL);
    if (!window) return EXIT_FAILURE;
    
    // create vertex buffer
    sfVertexArray* vertex_array = sfVertexArray_create();
    sfVertexArray_setPrimitiveType(vertex_array, sfPoints);
    for (int i = 0; i < GSIZE*GSIZE; ++i) {
        sfVertex vertex;
        vertex.color = sfBlack;
        vertex.position = (sfVector2f){(float) i / GSIZE, (float) (i % GSIZE)};
        sfVertexArray_append(vertex_array, vertex);
    }

    hipfftComplex* tmp = (hipfftComplex*) malloc(sizeof(hipfftComplex)*GSIZE*GSIZE);

    //initialize kernel
    hipfftReal* kgrid = get_kernel_in_grid(KSIZE, GSIZE);
    hipfftReal* ggrid = get_u10_grid(GSIZE, seed);
    
    hipfftReal*    device_neigh     = NULL;
    hipfftReal*    device_kgrid     = NULL;
    hipfftReal*    device_ggrid     = NULL;
    hipfftComplex* device_kgrid_fft = NULL;
    hipfftComplex* device_ggrid_fft = NULL;

    hipMalloc(&device_neigh     ,sizeof(hipfftReal)    * GSIZE * GSIZE);
    hipMalloc(&device_kgrid     ,sizeof(hipfftReal)    * GSIZE * GSIZE);
    hipMalloc(&device_ggrid     ,sizeof(hipfftReal)    * GSIZE * GSIZE);
    hipMalloc(&device_kgrid_fft ,sizeof(hipfftComplex) * GSIZE * GSIZE);
    hipMalloc(&device_ggrid_fft ,sizeof(hipfftComplex) * GSIZE * GSIZE);

    hipMemcpy(device_kgrid, kgrid, sizeof(hipfftReal) * GSIZE * GSIZE, hipMemcpyHostToDevice);
    hipMemcpy(device_ggrid, ggrid, sizeof(hipfftReal) * GSIZE * GSIZE, hipMemcpyHostToDevice);

    device_rfft2(device_kgrid, device_kgrid_fft, GSIZE);
    
    float avg_clock = 0;
    for (int n = 0;; ++n) {
        clock_t start = clock();

        device_rfft2(device_ggrid, device_ggrid_fft, GSIZE);
        scaled_hadamart_product<<<dim3(GSIZE),dim3(GSIZE)>>>(device_ggrid_fft, device_kgrid_fft, GSIZE);
        device_irfft2(device_ggrid_fft, device_neigh, GSIZE); 
        bosco_growth<<<dim3(GSIZE),dim3(GSIZE)>>>(device_ggrid, device_neigh, BOSCO_B1, BOSCO_B2, BOSCO_S1, BOSCO_S2, GSIZE);

        for (int i = 0; i < GSIZE*GSIZE; ++i) {
            sfVertex* vertex = sfVertexArray_getVertex(vertex_array, i);
            vertex->color = round(ggrid[i]) ? sfWhite : sfBlack;
        }

        sfRenderWindow_drawVertexArray(window, vertex_array, NULL);
        sfRenderWindow_display(window);

        hipMemcpy(ggrid, device_ggrid, sizeof(hipfftReal)*GSIZE*GSIZE, hipMemcpyDeviceToHost);

        // take time
        int msec = ((clock() - start) * 1000 / CLOCKS_PER_SEC)%1000;
        if(n > 100) avg_clock = (msec + (n-100) * avg_clock) / (n+1-100);
        printf("\rmsec: %d, avg:%f.", msec, avg_clock);
        fflush(stdout);

    }

    hipFree(device_neigh);
    hipFree(device_ggrid);
    hipFree(device_kgrid);
    hipFree(device_kgrid_fft);
    hipFree(device_ggrid_fft);

    free(kgrid);
    free(ggrid);
    free(tmp);
    return 0;
}
