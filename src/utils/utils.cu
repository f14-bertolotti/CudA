
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

__global__ void scaled_hadamart_product(hipfftComplex* A, hipfftComplex* B, int scale, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) {
        float x = (A[i].x*B[i].x - A[i].y*B[i].y);
        float y = (A[i].x*B[i].y + A[i].y*B[i].x);
        A[i].x = x/scale;
        A[i].y = y/scale;
    }
}

__global__ void game_of_life_growth(hipfftReal* grid, hipfftReal* neigh, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) {
        int n = round(neigh[i]);
        int c = round(grid[i]);
        grid[i] = max(0, min(1, c + (n == 3) - ((n < 2) || (n > 3))));
    }
}

__global__ void larger_than_life_growth(hipfftReal* grid, hipfftReal* neigh, int b1, int b2, int s1, int s2, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) {
        int c = grid[i];
        int n = neigh[i];
        grid[i] = max(0,min(1, c + ((n >= b1) & (n <= b2)) - ((n < s1) | (n > s2))));
    }
}

__global__ void primordia_growth(hipfftReal* grid, hipfftReal* neigh, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) {
        float c = grid[i];
        float n = neigh[i];
        grid[i] = max(0.0f,min(1.0f, c + (1/10.0f) * (((n >= 0.20)&(n <= 0.25)) - ((n <= 0.19)|(n >= 0.33)))));
    }
}
